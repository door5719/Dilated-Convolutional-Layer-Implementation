#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "hiprand.h"
#include "hipblas.h"

extern "C" {
#include "convolutional_layer.h"
#include "dilated_convolutional_layer.h"
#include "batchnorm_layer.h"
#include "gemm.h"
#include "blas.h"
#include "im2col_dilated.h"
#include "im2col.h"
#include "col2im.h"
#include "utils.h"
#include "hip/hip_runtime.h"
#include "darknet.h"
}

__global__ void binarize_kernel(float *x, int n, float *binary);


void binarize_gpu(float *x, int n, float *binary);

__global__ void binarize_input_kernel(float *input, int n, int size, float *binary);


void binarize_input_gpu(float *input, int n, int size, float *binary);



__global__ void binarize_weights_kernel(float *weights, int n, int size, float *binary);

void binarize_weights_gpu(float *weights, int n, int size, float *binary);


void forward_dilated_conv_layer_gpu(dilated_convolutional_layer l, network net)
{
    printf("I'm in forward_dilated_conv_layer_gpu!\n");
    fill_gpu(l.outputs*l.batch, 0, l.output_gpu, 1);
    printf("Fill GPU success!\n");
    if(l.binary){
        printf("Binarize in progress!\n");
        binarize_weights_gpu(l.weights_gpu, l.n, l.c/l.groups*l.size*l.size, l.binary_weights_gpu);
        swap_binary(&l);
    }

    if(l.xnor){
        printf("Xnor construction in progress!\n");
        binarize_weights_gpu(l.weights_gpu, l.n, l.c/l.groups*l.size*l.size, l.binary_weights_gpu);
        swap_binary(&l);
        binarize_gpu(net.input_gpu, l.c*l.h*l.w*l.batch, l.binary_input_gpu);
        net.input_gpu = l.binary_input_gpu;
    }

#ifdef CUDNN
    float one = 1;
    hipdnnConvolutionForward(cudnn_handle(),
                &one,
                l.srcTensorDesc,
                net.input_gpu,
                l.weightDesc,
                l.weights_gpu,
                l.convDesc,
                l.fw_algo,
                net.workspace,
                l.workspace_size,
                &one,
                l.dstTensorDesc,
                l.output_gpu);

#else
    int i, j;
    int m = l.n/l.groups;
    int k = l.size*l.size*l.c/l.groups;
    int n = l.out_w*l.out_h;
    for(i = 0; i < l.batch; ++i){
        for(j = 0; j < l.groups; ++j){
            float *a = l.weights_gpu + j*l.nweights/l.groups;
            float *b = net.workspace;
            float *c = l.output_gpu + (i*l.groups + j)*n*m;
            float *im = net.input_gpu + (i*l.groups + j)*l.c/l.groups*l.h*l.w;

            if (l.size == 1){
                b = im;
            } else {
                printf("I'm going to call im2col_dilated_gpu!\n");
                //-----------print im2col input-----------------------------------------------------
                /*printf("image = \n");
                float *temp = im;
                for (int i = 1; i <= l.inputs; i++)
                {
                    if (i % 10 == 0)
                    {
                        printf("%d\t", (int)*temp);
                        printf("\n");
                        temp = temp + 1;
                    }else{
                        printf("%d\t", (int)*temp);
                        temp = temp + 1;
                    }
                    //printf("i = %d\t", i);
                }*/
                //-----------------------------------------------------------------------------------
                im2col_dilated_gpu(im, l.c/l.groups, l.h, l.w, l.size, l.stride, l.pad, l.dilate_rate, b);
                //------------print im2col output-----------------------------------------------------
                printf("image_col = \n");
                float *temp = b;
                for (int i = 1; i <= 36*12; i++)
                {
                    if (i % 36 == 0)
                    {
                        printf("%d  ", (int)*temp);
                        printf("\n");
                        temp = temp + 1;
                    }else{
                        printf("%d  ", (int)*temp);
                        temp = temp + 1;
                    }
                }
                //-------------------------------------------------------------------------------------

            }
            //printf("I'm going to call gemm_gpu!\n");
            gemm_gpu(0,0,m,n,k,1,a,k,b,n,1,c,n);
            //printf("gemm_gpu finished\n");
            // TA = 0; TB = 0, M = m, N = n, K = k, Alpha = 1, *A = a, lda = n, *B = b, ldb = k, Beta = 1, *C = c, ldc = n
            /*void gemm_gpu(int TA, int TB, int M, int N, int K, float ALPHA, 
                float *A, int lda, 
                float *B, int ldb,
                float BETA,
                float *C, int ldc)*/

            // print gemm output
            /*printf("gemm_gpu output = \n");
            float *temp = c;
            for (int i = 1; i <= l.outputs; i++)
            {
                if (i % 2 == 0)
                {
                    printf("%f\t", *temp);
                    printf("\n");
                    temp = temp + 1;
                }else{
                    printf("%f\t", *temp);
                    temp = temp + 1;
            }*/
        }

    }
    
#endif

    if (l.batch_normalize) {
        forward_batchnorm_layer_gpu(l, net);
    } else {
        add_bias_gpu(l.output_gpu, l.biases_gpu, l.batch, l.n, l.out_w*l.out_h);
    }

    activate_array_gpu(l.output_gpu, l.outputs*l.batch, l.activation);
    //if(l.dot > 0) dot_error_gpu(l);
    if(l.binary || l.xnor) swap_binary(&l);
}

__global__ void smooth_kernel(float *x, int n, int w, int h, int c, int size, float rate, float *delta);


extern "C" void smooth_layer(layer l, int size, float rate);


void backward_dilated_conv_layer_gpu(convolutional_layer l, network net)
{
    if(l.smooth){
        smooth_layer(l, 5, l.smooth);
    }
    //constrain_gpu(l.outputs*l.batch, 1, l.delta_gpu, 1);
    gradient_array_gpu(l.output_gpu, l.outputs*l.batch, l.activation, l.delta_gpu);


    if(l.batch_normalize){
        backward_batchnorm_layer_gpu(l, net);
    } else {
        backward_bias_gpu(l.bias_updates_gpu, l.delta_gpu, l.batch, l.n, l.out_w*l.out_h);
    }
    float *original_input = net.input_gpu;

    if(l.xnor) net.input_gpu = l.binary_input_gpu;
#ifdef CUDNN
    float one = 1;
    hipdnnConvolutionBackwardFilter(cudnn_handle(),
            &one,
            l.srcTensorDesc,
            net.input_gpu,
            l.ddstTensorDesc,
            l.delta_gpu,
            l.convDesc,
            l.bf_algo,
            net.workspace,
            l.workspace_size,
            &one,
            l.dweightDesc,
            l.weight_updates_gpu);

    if(net.delta_gpu){
        if(l.binary || l.xnor) swap_binary(&l);
        hipdnnConvolutionBackwardData(cudnn_handle(),
                &one,
                l.weightDesc,
                l.weights_gpu,
                l.ddstTensorDesc,
                l.delta_gpu,
                l.convDesc,
                l.bd_algo,
                net.workspace,
                l.workspace_size,
                &one,
                l.dsrcTensorDesc,
                net.delta_gpu);
        if(l.binary || l.xnor) swap_binary(&l);
        if(l.xnor) gradient_array_gpu(original_input, l.batch*l.c*l.h*l.w, HARDTAN, net.delta_gpu);
    }

#else
    int m = l.n/l.groups;
    int n = l.size*l.size*l.c/l.groups;
    int k = l.out_w*l.out_h;

    int i, j;
    for(i = 0; i < l.batch; ++i){
        for(j = 0; j < l.groups; ++j){
            float *a = l.delta_gpu + (i*l.groups + j)*m*k;
            float *b = net.workspace;
            float *c = l.weight_updates_gpu + j*l.nweights/l.groups;

            float *im  = net.input_gpu+(i*l.groups + j)*l.c/l.groups*l.h*l.w;
            float *imd = net.delta_gpu+(i*l.groups + j)*l.c/l.groups*l.h*l.w;

            im2col_dilated_gpu(im, l.c/l.groups, l.h, l.w, l.size, l.stride, l.pad, 3, b);
            gemm_gpu(0,1,m,n,k,1,a,k,b,k,1,c,n);

            if (net.delta_gpu) {
                if (l.binary || l.xnor) swap_binary(&l);
                a = l.weights_gpu + j*l.nweights/l.groups;
                b = l.delta_gpu + (i*l.groups + j)*m*k;
                c = net.workspace;
                if (l.size == 1) {
                    c = imd;
                }

                gemm_gpu(1,0,n,k,m,1,a,n,b,k,0,c,k);

                if (l.size != 1) {
                    col2im_gpu(net.workspace, l.c/l.groups, l.h, l.w, l.size, l.stride, l.pad, imd);
                }
                if(l.binary || l.xnor) {
                    swap_binary(&l);
                }
            }
            if(l.xnor) gradient_array_gpu(original_input + i*l.c*l.h*l.w, l.c*l.h*l.w, HARDTAN, net.delta_gpu + i*l.c*l.h*l.w);
        }
    }
#endif
}

void pull_dilated_conv_layer(layer l)
{
    cuda_pull_array(l.weights_gpu, l.weights, l.nweights);
    cuda_pull_array(l.biases_gpu, l.biases, l.n);
    cuda_pull_array(l.weight_updates_gpu, l.weight_updates, l.nweights);
    cuda_pull_array(l.bias_updates_gpu, l.bias_updates, l.n);
    if (l.batch_normalize){
        cuda_pull_array(l.scales_gpu, l.scales, l.n);
        cuda_pull_array(l.rolling_mean_gpu, l.rolling_mean, l.n);
        cuda_pull_array(l.rolling_variance_gpu, l.rolling_variance, l.n);
    }
}

void push_dilated_conv_layer(layer l)
{
    cuda_push_array(l.weights_gpu, l.weights, l.nweights);
    cuda_push_array(l.biases_gpu, l.biases, l.n);
    cuda_push_array(l.weight_updates_gpu, l.weight_updates, l.nweights);
    cuda_push_array(l.bias_updates_gpu, l.bias_updates, l.n);
    if (l.batch_normalize){
        cuda_push_array(l.scales_gpu, l.scales, l.n);
        cuda_push_array(l.rolling_mean_gpu, l.rolling_mean, l.n);
        cuda_push_array(l.rolling_variance_gpu, l.rolling_variance, l.n);
    }
}


void update_dilated_conv_layer_gpu(layer l, update_args a)
{
    float learning_rate = a.learning_rate*l.learning_rate_scale;
    float momentum = a.momentum;
    float decay = a.decay;
    int batch = a.batch;

    if(a.adam){
        adam_update_gpu(l.weights_gpu, l.weight_updates_gpu, l.m_gpu, l.v_gpu, a.B1, a.B2, a.eps, decay, learning_rate, l.nweights, batch, a.t);
        adam_update_gpu(l.biases_gpu, l.bias_updates_gpu, l.bias_m_gpu, l.bias_v_gpu, a.B1, a.B2, a.eps, decay, learning_rate, l.n, batch, a.t);
        if(l.scales_gpu){
            adam_update_gpu(l.scales_gpu, l.scale_updates_gpu, l.scale_m_gpu, l.scale_v_gpu, a.B1, a.B2, a.eps, decay, learning_rate, l.n, batch, a.t);
        }
    }else{
        axpy_gpu(l.nweights, -decay*batch, l.weights_gpu, 1, l.weight_updates_gpu, 1);
        axpy_gpu(l.nweights, learning_rate/batch, l.weight_updates_gpu, 1, l.weights_gpu, 1);
        scal_gpu(l.nweights, momentum, l.weight_updates_gpu, 1);

        axpy_gpu(l.n, learning_rate/batch, l.bias_updates_gpu, 1, l.biases_gpu, 1);
        scal_gpu(l.n, momentum, l.bias_updates_gpu, 1);

        if(l.scales_gpu){
            axpy_gpu(l.n, learning_rate/batch, l.scale_updates_gpu, 1, l.scales_gpu, 1);
            scal_gpu(l.n, momentum, l.scale_updates_gpu, 1);
        }
    }
    if(l.clip){
        constrain_gpu(l.nweights, l.clip, l.weights_gpu, 1);
    }
}

void test_dilated_conv_layer_gpu()
{
    printf("Entering test_dilated_conv_layer()\n");
    dilated_convolutional_layer l = make_dilated_conv_layer(1, 10, 10, 3, 1, 1, 2, 1, 0, LEAKY, 0, 0, 0, 0, 2);
    // batch = 1, h = 10, w = 10, c = 3, n = 1, group = 1, size = 2, stride = 1, padding = 0, activation = LEAKY, 
    // batch_nomarlize = 0, binary = 0, xnor = 0, adam = 0, dilate_rate = 2
    printf("make dilated conv layer success!\n");
    float data[] = {
        1,1,1,1,1,1,1,1,1,1,
        2,2,2,2,2,2,2,2,2,2,
        3,3,3,3,3,3,3,3,3,3,
        4,4,4,4,4,4,4,4,4,4,
        5,5,5,5,5,5,5,5,5,5,
        6,6,6,6,6,6,6,6,6,6,
        7,7,7,7,7,7,7,7,7,7,
        8,8,8,8,8,8,8,8,8,8,
        9,9,9,9,9,9,9,9,9,9,
        9,9,9,9,9,9,9,9,9,9,

        1,1,1,1,1,1,1,1,1,1,
        2,2,2,2,2,2,2,2,2,2,
        3,3,3,3,3,3,3,3,3,3,
        4,4,4,4,4,4,4,4,4,4,
        5,5,5,5,5,5,5,5,5,5,
        6,6,6,6,6,6,6,6,6,6,
        7,7,7,7,7,7,7,7,7,7,
        8,8,8,8,8,8,8,8,8,8,
        9,9,9,9,9,9,9,9,9,9,
        9,9,9,9,9,9,9,9,9,9,

        1,1,1,1,1,1,1,1,1,1,
        2,2,2,2,2,2,2,2,2,2,
        3,3,3,3,3,3,3,3,3,3,
        4,4,4,4,4,4,4,4,4,4,
        5,5,5,5,5,5,5,5,5,5,
        6,6,6,6,6,6,6,6,6,6,
        7,7,7,7,7,7,7,7,7,7,
        8,8,8,8,8,8,8,8,8,8,
        9,9,9,9,9,9,9,9,9,9,
        9,9,9,9,9,9,9,9,9,9};

    float w[] = {
        1,1,1,1,1,1,1,1,1,1,1,1
    };
    float out[] = {0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0};
    network net = *make_network(1);
    net.layers = &l;
    net.input_gpu = data;
    net.workspace = (float*) calloc(1, l.outputs);
    l.weights_gpu = w;
    forward_dilated_conv_layer_gpu(l, net);
    
    //float *temp = out;
    //hipMemcpy(temp, l.output_gpu, l.outputs*sizeof(float),hipMemcpyDeviceToHost);
    //printf("Output:\n");
    //printf("Number of output: %d\n", l.outputs);
    /*for (int i = 1; i <= l.outputs; i++)
    {
        if (i % 6 == 0)
        {
            printf("%f\t", *temp);
            printf("\n");
            temp = temp + 1;
        }else{
            printf("%f\t", *temp);
            temp = temp + 1;
        }
        //printf("i = %d\t", i);
    }*/
}

